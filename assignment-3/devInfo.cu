#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    hipDeviceProp_t props;
    int devCount;
    hipGetDeviceCount(&devCount);
    
    for(int i = 0; i < devCount; ++i)
    {
            hipGetDeviceProperties(&props, 0);
            printf("%d%d\n", props.major, props.minor);
    }
}
