#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "../utils/utils.h"
//TODO: debloat

/* gets last cuda error and if it's not a hipSuccess
 * prints debug information on stderr and aborts */
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            exit(1); \
        } \
    } while (0)

/* read a table from a file */
//TODO: move in generic functions file.
void read_from_file(int *X, char *filename, int N)
{
    FILE *fp = fopen(filename, "r+");
    int size = fread(X, sizeof(int), N * N, fp);
#ifdef TEST
    printf("total elements: %d\n", size);
#endif  // TEST
    fclose(fp);
}

//TODO: move in generic functions file.
void save_table(int *X, int N)
{
    FILE *fp;
    char filename[20];
    sprintf(filename, "cuda-results.bin");
#ifdef TEST
    printf("Saving table in file %s\n", filename);
#endif  // TEST
    fp = fopen(filename, "w+");
    fwrite(X, sizeof(int), N * N, fp);
    fclose(fp);
}

//TODO: change it with nvidia's function
/* Determines the number of threads per block.
 * Returns a power of 2 number that evenly divides the total number of elements*/
int find_thread_count(const int dim)
{
    if (dim == 0) return 0;
    int result = 2;
    while ((dim % result == 0) && (result < 1024)) result *= 2;
    return result >> 1;
}

__global__ void cuda_compute(int *d_help, const int *d_table, int N)
{
    const int cell_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = cell_id % N;
    const int i = (cell_id - j) / N;

    const int left = (i-1+N)%N;
    const int right = (i+1)%N;
    const int up = (j-1+N)%N;
    const int down = (j+1)%N;

    const int alive_neighbors = d_table[POS(left , up  )] +
                                d_table[POS(left , j   )] +
                                d_table[POS(left , down)] +
                                d_table[POS(i    , up  )] +
                                d_table[POS(i    , down)] +
                                d_table[POS(right, up  )] +
                                d_table[POS(right, j   )] +
                                d_table[POS(right, down)] ;
    if (cell_id < N * N)
        d_help[cell_id] = (alive_neighbors == 3) || (alive_neighbors == 2 && d_table[cell_id]) ? 1 : 0;
}

//TODO: move in generic functions file.
void print_table(int* A, int N)
{
    for (int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%s%d "ANSI_COLOR_RESET, A[i * N + j] ? ANSI_COLOR_BLUE : ANSI_COLOR_RED, A[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char **argv)
{
    if (argc < 3) {
        printf("usage: %s FILE dimension\n", argv[0]);
        exit(1);
    }

    int n_runs;
    if (argc == 4) n_runs = atoi(argv[3]);
    else n_runs = DFL_RUNS;

    const int N = atoi(argv[2]);
    const int total_elements = N * N;
    const int mem_size = total_elements * sizeof(int);

    char* filename = argv[1];
    int *table;
    printf("Reading %dx%d table from file %s\n", N, N, filename);
    table = (int*) malloc(mem_size);
    read_from_file(table, filename, N);
    printf("Finished reading table\n");

#ifdef PRINT
    print_table(table, N);
#endif

    int t_count = find_thread_count(total_elements);
    dim3 thread_count(t_count);
    //TODO: fix error with blocks count when the input array is big
    dim3 blocks_count(total_elements / t_count);

    int *d_help, *d_table;
    hipMalloc((void **) &d_help,  mem_size);
    cudaCheckErrors("malloc fail");

    hipMalloc((void **) &d_table, mem_size);
    cudaCheckErrors("malloc fail");

    hipMemcpy(d_table, table, mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("memcpy fail");

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    for (int i = 0; i < n_runs; ++i) {
    cuda_compute <<< blocks_count, thread_count >>>(d_help, d_table, N);
        cudaCheckErrors("compute fail");
        swap(&d_table, &d_help);

#ifdef PRINT
        hipMemcpy(table, d_table, mem_size, hipMemcpyDeviceToHost);
        print_table(table, N);
#endif
    }

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;
    printf("Time to run:  %3.1f ms \n", time);

    hipMemcpy(table, d_table, total_elements * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceReset();
    save_table(table, N);
}
