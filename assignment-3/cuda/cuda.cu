#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <assert.h>
//TODO: debloat

void swap(int** a, int** b);

/* define colors */
//TODO: move them somewhere better
#define ANSI_COLOR_RED     "\x1b[31m"
#define ANSI_COLOR_GREEN   "\x1b[32m"
#define ANSI_COLOR_YELLOW  "\x1b[33m"
#define ANSI_COLOR_BLUE    "\x1b[34m"
#define ANSI_COLOR_MAGENTA "\x1b[35m"
#define ANSI_COLOR_CYAN    "\x1b[36m"
#define ANSI_COLOR_RESET   "\x1b[0m"

/* gets last cuda error and if it's not a cudaSuccess
 * prints debug information on stderr and aborts */
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            exit(1); \
        } \
    } while (0)


/* table is where we store the actual data,
 * help_table is used for the calculation of a new generation */
int *table;
int *help_table;
unsigned int N;

#define DFL_RUNS 10

/* swap 2 int* pointers */
//TODO: move somewhere better, make it a #define(?)
inline void swap(int** a, int** b)
{
    int *t;
    t = *a;
    *a = *b;
    *b = t;
}

/* read a table from a file */
//TODO: move it somewhere better, seperate file
void read_from_file(int *X, char *filename, int N)
{
    FILE *fp = fopen(filename, "r+");
    int size = fread(X, sizeof(int), N * N, fp);
#ifdef TEST
    printf("total elements: %d\n", size);
#endif  // TEST
    fclose(fp);
}

void save_table(int *X, int N)
{
    FILE *fp;
    char filename[20];
    sprintf(filename, "results.bin");
#ifdef TEST
    printf("Saving table in file %s\n", filename);
#endif  // TEST
    fp = fopen(filename, "w+");
    fwrite(X, sizeof(int), N * N, fp);
    fclose(fp);
}

#define POS(i, j) (i*N + j)
#define cPOS(i, j) (i*Nc + j)

void pre_calc(int* prev_of, int* next_of)
{

    prev_of[0] = N - 1;
    next_of[N - 1] = 0;
    for (int i = 1; i < N; ++i) prev_of[i] = i - 1;
    for (int i = 0; i < N - 1; ++i) next_of[i] = i + 1;
}

int find_thread_count(const int dim)
{
    if (dim == 0) return 0;
    int result = 2;
    while (dim % result == 0) result *= 2;
    return result >> 1;
}

__global__ void cuda_compute(int *d_help, const int *d_table, const int *prev, const int *next, unsigned int Nc)
{
    const unsigned int cell_id = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int i = cell_id / Nc;
    const unsigned int j = cell_id % Nc;

    const int left = prev[i];
    const int right = next[i];

    const int up = prev[j];
    const int down = next[j];

    const int alive_neighbors = d_table[cPOS(left , up  )] +
                                d_table[cPOS(left , j   )] +
                                d_table[cPOS(left , down)] +
                                d_table[cPOS(i    , up  )] +
                                d_table[cPOS(i    , down)] +
                                d_table[cPOS(right, up  )] +
                                d_table[cPOS(right, j   )] +
                                d_table[cPOS(right, down)] ;
    if (cell_id < Nc * Nc)
        d_help[cell_id] = (alive_neighbors == 3) || (alive_neighbors == 2 && d_table[cell_id]) ? 1 : 0;
}

void print_table(int* A)
{
    for (int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%s%d "ANSI_COLOR_RESET, A[i * N + j] ? ANSI_COLOR_BLUE : ANSI_COLOR_RED, A[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char **argv)
{
    if (argc < 3) {
        printf("usage: %s FILE dimension\n", argv[0]);
        exit(1);
    }

    int N_RUNS = DFL_RUNS;
    char* filename = argv[1];
    N = atoi(argv[2]);
    const int total_size = N * N;
    const int mem_size = total_size * sizeof(int);
    const int row_mem_size = N * sizeof(int);
    if (argc == 4) {
        N_RUNS = atoi(argv[3]);
    }

    printf("Reading %dx%d table from file %s\n", N, N, filename);
    table = (int*) malloc(mem_size);
    help_table = (int*) malloc(mem_size);
    read_from_file(table, filename, N);
    printf("Finished reading table\n");

#ifdef PRINT
    print_table(table);
#endif

    int *d_help, *d_table, *prev, *next;

    dim3 grid(N, N);
    const int thread_count = find_thread_count(total_size);
    const int blocks_count = total_size / thread_count;

    int *prev_of;
    int *next_of;
    prev_of = (int*) malloc(N * sizeof(size_t));
    next_of = (int*) malloc(N * sizeof(size_t));
    pre_calc(prev_of, next_of);

    hipMalloc((void **) &d_help,  mem_size);
    cudaCheckErrors("malloc fail");

    hipMalloc((void **) &d_table, mem_size);
    cudaCheckErrors("malloc fail");

    hipMalloc((void **) &prev, row_mem_size);
    cudaCheckErrors("malloc fail");

    hipMalloc((void **) &next, row_mem_size);
    cudaCheckErrors("malloc fail");

    hipMemcpy(d_help, help_table, mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("memcpy fail");

    hipMemcpy(d_table, table, mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("memcpy fail");

    hipMemcpy(prev, prev_of, row_mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("memcpy fail");

    hipMemcpy(next, next_of, row_mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("memcpy fail");
    
    struct timeval startwtime, endwtime;
    gettimeofday (&startwtime, NULL);
    
    for (int i = 0; i < N_RUNS; ++i) {
        cuda_compute <<< blocks_count, thread_count >>>(d_help, d_table, prev, next, N);
        cudaCheckErrors("compute fail");
        swap(&d_table, &d_help);
        
#ifdef PRINT
        cudaMemcpy(table, d_table, mem_size, cudaMemcpyDeviceToHost);
        print_table(table);
#endif
    }

    gettimeofday (&endwtime, NULL);
    double exec_time = (double)((endwtime.tv_usec - startwtime.tv_usec)
                                / 1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
    printf("clock: %fs\n", exec_time);

    hipMemcpy(table, d_table, total_size * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceReset();
    save_table(table, N);

    free(table);
    free(help_table);
}
