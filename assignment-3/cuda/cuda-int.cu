#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "../utils/utils.h"

/**
 * @brief Gets last cuda error and if it's not a hipSuccess
 * prints debug information on stderr and aborts.
 */
#define cudaCheckErrors(msg, yolo, yolo2) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
              msg, hipGetErrorString(__err), \
              __FILE__, __LINE__); \
      exit(1); \
    } \
  } while (0)

/**
 * @brief The width of a tile assigned to a thread.
 */
#define CONF_WIDTH 8
/**
 * @brief The height of a tile assigned to a thread.
 */
#define CONF_HEIGHT 4

// TODO: a conversion between the two grid of life representations
// int per cell <-> uint per 32 cells in the following configuration
//  0  1  2  3  4  5  6  7
//  8  9 10 11 12 13 14 15
// 16 17 18 19 20 21 22 23
// 24 25 26 27 28 29 30 31
// width of this tile: 8
// height of this tile: 4

// TODO: Test if implementation is correct.

/**
 * @brief Kernel that advances the GOL by one generation.
 * @returns Nothing.
 * @param d_table The GOL matrix of the current generation.
 * @param d_result The resulting GOL matrix of the next generation.
 * @param m_height
 * @param m_width
 * @param m_size
 */
__global__ void calculate_next_generation(
  uint const *d_table, uint *d_result,
  uint m_height, uint m_width, uint m_size)
{
  const int row = (__mul24(blockIdx.x, blockDim.x) + threadIdx.x) * m_width;
  const int col = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;

  if (row + col >= m_size) return;

  const int t_row = (row - m_width + m_size) % m_size;
  const int b_row = (row + m_width) % m_size;
  const int l_col = (col - 1 + m_width) % m_width;
  const int r_col = (col + 1) % m_width;

  // ~ #ifdef PRINT
  // ~ printf("Thread %d - %d:\n"
  // ~ "top row: %d, bottom row: %d\n"
  // ~ "left col: %d, right col: %d\n", row, col, t_row, b_row, l_col, r_col);
  // ~ #endif  // PRINT

  //TODO: write only own tile to shared memory, write some edges of the block in shared memory and then sync and read neighbors from shared memory
  // bring information to local memory (global/cache/registers)
  const uint this_tile = d_table[row + col    ];
  const uint tl_tile = d_table  [t_row + l_col];
  const uint t_tile = d_table   [t_row + col  ];
  const uint tr_tile = d_table  [t_row + r_col];
  const uint l_tile = d_table   [row + l_col  ];
  const uint r_tile = d_table   [row + r_col  ];
  const uint bl_tile = d_table  [b_row + l_col];
  const uint b_tile = d_table   [b_row + col  ];
  const uint br_tile = d_table  [b_row + r_col];

  // ~ #ifdef PRINT
  // ~ printf("Thread %d-%d:\n"
  // ~ "tl: %X, t: %X, tr: %X\n"
  // ~ "l: %X, this: %X, r: %X\n"
  // ~ "bl: %X, b: %X, br: %X\n", row, col,
  // ~ tl_tile, t_tile, tr_tile,
  // ~ l_tile, this_tile, r_tile,
  // ~ bl_tile, b_tile, br_tile);
  // ~ #endif  // PRINT


  // build resulting tile in local memory (register)
  uint result_tile = 0;
  uint alive_cells;
  uint first_cells, second_cells;

  // if we represent a 8x4 array A as int then we can access position [i,j] of the array like this:
  //(A >> (i + 8 * j) & 1u)

  // Update vertical edge 1 - 6
  first_cells = (this_tile & 1u) +
                (this_tile >> 8 & 1u) +
                (t_tile >> 24 & 1u);
  second_cells = (this_tile >> 1 & 1u) +
                 (this_tile >> 9 & 1u) +
                 (t_tile >> 25 & 1u);
  //TODO: IDEA: replace (x >> p & 1) with ((x & (2**p)) != 0)
  //TODO: pragma unroll probably doesn't cause any problems. reenable it after code works correctly.
  // ~ #pragma unroll

  //TODO: IDEA: instead of having an if statement inside the loop have first_cells and seconds_cells in an array[2]
  for (int i = 1; i < 7; ++i) {
    uint this_cell = this_tile >> i & 1u;

    // (x & 1u) == (x % 2) , x >= 0 but mod operator is relatively slow in cuda so we avoid it.
    if (i & 1u) {
      alive_cells = first_cells;
      first_cells = (this_tile >> (i + 1) & 1u) +
                    (this_tile >> (i + 9) & 1u) +
                    (t_tile >> (i + 25) & 1u);
      alive_cells += first_cells;
      alive_cells += second_cells - this_cell;
    } else {
      alive_cells = second_cells;
      second_cells = (this_tile >> (i + 1) & 1u) +
                     (this_tile >> (i + 9) & 1u) +
                     (t_tile >> (i + 25) & 1u);
      alive_cells += second_cells;
      alive_cells += first_cells - this_cell;
    }

    //TODO: replace and profile with:
    // ~ result_tile |= ((alive_cells == 3) || (alive_cells == 2 && this_cell)) * (1u << i);
    result_tile |= (alive_cells == 3) || (alive_cells == 2
                                          && this_cell) ? (1u << i) : 0u;
  }

  // Update 9 - 14
  first_cells = (this_tile & 1u) +
                (this_tile >> 8 & 1u) +
                (this_tile >> 16 & 1u);
  second_cells = (this_tile >> 1 & 1u) +
                 (this_tile >> 9 & 1u) +
                 (this_tile >> 17 & 1u);

  // ~ #pragma unroll

  for (int i = 9; i < 15; ++i) {
    uint this_cell = (this_tile >> i) & 1u;

    if (i & 1u) {
      alive_cells = first_cells;
      first_cells = (this_tile >> (i + 1) & 1u) +
                    (this_tile >> (i + 9) & 1u) +
                    (this_tile >> (i - 7) & 1u);
      alive_cells += first_cells;
      alive_cells += second_cells - this_cell;
    } else {
      alive_cells = second_cells;
      second_cells = (this_tile >> (i + 1) & 1u) +
                     (this_tile >> (i + 9) & 1u) +
                     (this_tile >> (i - 7) & 1u);
      alive_cells += second_cells;
      alive_cells += first_cells - this_cell;
    }

    result_tile |= (alive_cells == 3) || (alive_cells == 2
                                          && this_cell) ? (1u << i) : 0u;
  }

  // Update 17 - 22
  first_cells = (this_tile >> 24 & 1u) +
                (this_tile >> 8 & 1u) +
                (this_tile >> 16 & 1u);
  second_cells = (this_tile >> 25 & 1u) +
                 (this_tile >> 9 & 1u) +
                 (this_tile >> 17 & 1u);

  // ~ #pragma unroll

  for (int i = 17; i < 23; ++i) {
    uint this_cell = (this_tile >> i) & 1u;

    if (i & 1u) {
      alive_cells = first_cells;
      first_cells = (this_tile >> (i + 1) & 1u) +
                    (this_tile >> (i + 9) & 1u) +
                    (this_tile >> (i - 7) & 1u);
      alive_cells += first_cells;
      alive_cells += second_cells - this_cell;
    } else {
      alive_cells = second_cells;
      second_cells = (this_tile >> (i + 1) & 1u) +
                     (this_tile >> (i + 9) & 1u) +
                     (this_tile >> (i - 7) & 1u);
      alive_cells += second_cells;
      alive_cells += first_cells - this_cell;
    }

    result_tile |= (alive_cells == 3) || (alive_cells == 2
                                          && this_cell) ? (1u << i) : 0u;
  }

  // Update vertical edge 25 - 30
  first_cells = (this_tile >> 24 & 1u) +
                (b_tile & 1u) +
                (this_tile >> 16 & 1u);
  second_cells = (this_tile >> 25 & 1u) +
                 (b_tile >> 1 & 1u) +
                 (this_tile >> 17 & 1u);

  // ~ #pragma unroll

  for (int i = 25; i < 31; ++i) {
    uint this_cell = this_tile >> i & 1u;

    if (i & 1u) {
      alive_cells = first_cells;
      first_cells = (this_tile >> (i - 7) & 1u) +
                    (this_tile >> (i + 1) & 1u) +
                    (b_tile >> (i - 23) & 1u);
      alive_cells += first_cells;
      alive_cells += second_cells - this_cell;
    } else {
      alive_cells = second_cells;
      second_cells = (this_tile >> (i - 7) & 1u) +
                     (this_tile >> (i + 1) & 1u) +
                     (b_tile >> (i - 23) & 1u);
      alive_cells += second_cells;
      alive_cells += first_cells - this_cell;
    }

    result_tile |= (alive_cells == 3) || (alive_cells == 2
                                          && this_cell) ? (1u << i) : 0u;
  }

  // Update corners 0, 7, 24, 31
  // Update 0. Needs t, tl, l.
  //TODO: use ILP? http://en.wikipedia.org/wiki/Instruction-level_parallelism
  alive_cells =
    (tl_tile >> 31) +
    (t_tile >> 24 & 1u) +
    (t_tile >> 25 & 1u) +
    (this_tile >> 1 & 1u) +
    (this_tile >> 9 & 1u) +
    (this_tile >> 8 & 1u) +
    (l_tile >> 7 & 1u) +
    (l_tile >> 15 & 1u);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile & 1u)) ? 1u : 0u;
  alive_cells =
    (tr_tile >> 24 & 1u) +
    (t_tile >> 30 & 1u) +
    (t_tile >> 31) +
    (this_tile >> 6 & 1u) +
    (this_tile >> 14 & 1u) +
    (this_tile >> 15 & 1u) +
    (r_tile & 1u) +
    (r_tile >> 8 & 1u);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile >> 7 & 1u)) ? (1u << 7) : 0u;
  alive_cells =
    (bl_tile >> 7 & 1u) +
    (b_tile & 1u) +
    (b_tile >> 1 & 1u) +
    (this_tile >> 16 & 1u) +
    (this_tile >> 17 & 1u) +
    (this_tile >> 25 & 1u) +
    (l_tile >> 23 & 1u) +
    (l_tile >> 31);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile >> 24 & 1u)) ? (1u << 24) : 0u;
  alive_cells =
    (br_tile & 1u) +
    (b_tile >> 6 & 1u) + (b_tile >> 7 & 1u) +
    (this_tile >> 22 & 1u) + (this_tile >> 23 & 1u) + (this_tile >> 30 & 1u) +
    (r_tile >> 16 & 1u) + (r_tile >> 24 & 1u);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile >> 31)) ? (1u << 31) : 0u;

  // Update vertical edges 8, 16, 15, 23
  alive_cells =
    (this_tile & 1u) +
    (this_tile >> 16 & 1u) +
    (this_tile >> 1 & 1u) +
    (this_tile >> 9 & 1u) +
    (this_tile >> 17 & 1u) +
    (l_tile >> 7 & 1u) +
    (l_tile >> 15 & 1u) +
    (l_tile >> 23 & 1u);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile >> 8 & 1u)) ? (1u << 8) : 0u;
  alive_cells =
    (this_tile >> 8 & 1u) +
    (this_tile >> 24 & 1u) +
    (this_tile >> 9 & 1u) +
    (this_tile >> 17 & 1u) +
    (this_tile >> 25 & 1u) +
    (l_tile >> 31) +
    (l_tile >> 15 & 1u) +
    (l_tile >> 23 & 1u);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile >> 16 & 1u)) ? (1u << 16) : 0u;
  alive_cells =
    (this_tile >> 7 & 1u) +
    (this_tile >> 23 & 1u) +
    (this_tile >> 6 & 1u) +
    (this_tile >> 14 & 1u) +
    (this_tile >> 22 & 1u) +
    (r_tile & 1u) +
    (r_tile >> 8 & 1u) +
    (r_tile >> 16 & 1u);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile >> 15 & 1u)) ? (1u << 15) : 0u;
  alive_cells =
    (this_tile >> 15 & 1u) +
    (this_tile >> 31) +
    (this_tile >> 30 & 1u) +
    (this_tile >> 14 & 1u) +
    (this_tile >> 22 & 1u) +
    (r_tile >> 24 & 1u) +
    (r_tile >> 8 & 1u) +
    (r_tile >> 16 & 1u);
  result_tile |= (alive_cells == 3) || (alive_cells == 2
                                        && (this_tile >> 23 & 1u)) ? (1u << 23) : 0u;

  // send result but to global memory
  d_result[row + col] = result_tile;
}

/**
 * @brief Creates a tiled GOL matrix from a normal GOL matrix.
 * @returns Nothing.
 * @param d_table The normal GOL matrix.
 * @param d_utable The output tiled GOL matrix
 * @param m_height
 * @param m_width
 * @param m_size
 */
__global__ void convert_to_tiled(
  int const *d_table, uint *d_utable,
  uint m_height, uint m_width, uint m_size)
{
  const int row = (__mul24(blockIdx.x, blockDim.x) + threadIdx.x) * m_width;
  const int col = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;

  if (row + col >= m_size) return;

  const int start_i = row * CONF_WIDTH * CONF_HEIGHT;
  const int start_j = col * CONF_WIDTH;
  uint place = 1u;
  uint tile = 0u;

  const int step_i = m_width * CONF_WIDTH;
  const int end_i = start_i + CONF_HEIGHT * step_i;
  const int end_j = start_j + CONF_WIDTH;

  for (int i = start_i; i < end_i; i += step_i) {
#pragma unroll

    for (int j = start_j; j < end_j; ++j) {
      tile |= place * d_table[j + i];
      place <<= 1;
    }
  }

  d_utable[col + row] = tile;
}

/**
 * @brief Creates a normal GOL matrix from a tiled GOL matrix.
 * @returns Nothing.
 * @param d_table The output normal GOL matrix.
 * @param d_utable The tiled GOL matrix
 * @param m_height
 * @param m_width
 * @param m_size
 */
__global__ void convert_from_tiled(
  int *d_table, uint const *d_utable,
  uint m_height, uint m_width, uint m_size)
{
  int row = (__mul24(blockIdx.x, blockDim.x) + threadIdx.x) * m_width;
  int col = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;

  if (row + col >= m_size) return;

  int start_i = row * CONF_WIDTH * CONF_HEIGHT;
  int start_j = col * CONF_WIDTH;
  int place = 0;

  const uint tile = d_utable[col + row];

  const int step_i = m_width * CONF_WIDTH;
  const int end_i = start_i + CONF_HEIGHT * step_i;
  const int end_j = start_j + CONF_WIDTH;
  int i, j;

  for (i = start_i; i < end_i; i += step_i) {
    for (j = start_j; j < end_j; ++j)
      d_table[j + i] = (int) (tile >> place++ & 1u);
  }
}

#ifndef TESTING

/**
 * @brief Main function
 * */
int main(int argc, char **argv)
{
  /******************************************************************************
   *                    Initialization of program variables                     *
   ******************************************************************************/

  if (argc < 3) {
    printf("usage: %s filename dimension (iterations)\n", argv[0]);
    exit(1);
  }

  int n_runs;

  // get number of GOL generations if available else use the default
  if (argc >= 4) n_runs = atoi(argv[3]);
  else n_runs = DFL_RUNS;

  /* The dimension of one side of the GOL square matrix. */
  const uint dim = atoi(argv[2]);
  /* Total cells in the GOL square matrix. */
  const uint total_elements = dim * dim;
  /* Size of the GOL matrix in bytes. */
  const uint mem_size = total_elements * sizeof(int);
  /* The width of a tile assigned to a thread. */
  const uint thread_width = CONF_WIDTH;
  /* The height of a tile assigned to a thread. */
  const uint thread_height = CONF_HEIGHT;
  /* Example for a 8x4 tile:
   * cuda-int implementation:
   * 32 cells in sizeof(int) bytes = 4 bytes = 32 bits => 1 cell : 1 bit
   * simplistic implementation:
   * 1 : 4 bytes = 32 bits
   *
   * cuda-int implementation is 32 times smaller in memory!
   * speed-up is also achieved because of less movement across global, cache,
   * register and shared memories and because registers and operations in
   * single-precision GPUs are using 32 bits.
   **/
  /* Total cells in the tiled GOL matrix. */
  const uint total_elements_tiled = total_elements / (thread_height * thread_width);
  /* The total size of the tiled GOL matrix in bytes. */
  const uint mem_size_tiled = mem_size / (thread_height * thread_width);
  /* Number of tiles in width. */
  const uint m_width = dim / thread_width;
  /* Number of tiles in height. */
  const uint m_height = dim / thread_height;
  // get name of file which contains the initial GOL matrix


  // ~ int blockSize;
  // ~ int minGridSize;
  // ~ hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, calculate_next_generation);
  // ~ printf("%d %d\n", blockSize, minGridSize);

  // Warning! grid and block sizes that correspond to a bigger array will cause leaks
  // these leaks in convert_to and convert_from are currently harmless (no failure)
  // dim == 1000 == 8 * 5 * 25
  // dim == 1000 == 4 * 10 * 25
  // x > y (?)
  dim3 block(1, 1);
  dim3 grid(1, 1);

  if (argc >= 8) {
    block = dim3(atoi(argv[4]), atoi(argv[5]));
    grid = dim3(atoi(argv[6]), atoi(argv[7]));
  }
  else {
    uint x = 32u;
    uint y = 32u;
    do {
      x >>= 1;
    } while(x >= m_height);
    do {
      y >>= 1;
    } while(y >= m_width);
    block = dim3(x, y)
    grid = dim3((int)(ceil(m_height/(float)x)), (int)(ceil(m_width/(float)y)));
  }

  char *filename = argv[1];
  // initialize and parse the matrix out of the file
  int *table;
  printf("Reading %dx%d table from file %s\n", dim, dim, filename);
  table = (int *) malloc(mem_size);
  read_from_file(table, filename, dim, dim);
  printf("Finished reading table\n");
#ifdef PRINT
  print_table(table, dim, dim);
#endif  // PRINT

  printf("%s: Running on a grid(%d, %d) with a block(%d, %d):\nFilename: %s with dim %d for %d iterations\n", argv[0], grid.x, grid.y, block.x, block.y, filename, dim, n_runs);

  /******************************************************************************
   *                              Table Conversion                              *
   ******************************************************************************/

  /******************************************************************************
   *                           Device initialization                            *
   ******************************************************************************/

  // Allocate memory on device.
  uint *d_tiled_table; /* Tiled matrix in device memory. */
  int *d_table; /* Original GOl matrix in device memory. */
  hipMalloc((void **) &d_table,  mem_size);
  cudaCheckErrors("device allocation of GOL matrix failed", __FILE__, __LINE__);
  hipMalloc((void **) &d_tiled_table, mem_size_tiled);
  cudaCheckErrors("device allocation of GOL uint tiled matrix failed", __FILE__, __LINE__);

  // Transfer memory from initial matrix from host to device.
  hipMemcpy(d_table, table, mem_size, hipMemcpyHostToDevice);
  cudaCheckErrors("copy from host to device memory failed", __FILE__, __LINE__);

  convert_to_tiled <<< grid, block >>>(d_table, d_tiled_table,
                                       m_height, m_width, total_elements_tiled);
  cudaCheckErrors("failed to convert normal repr to uint tiled repr", __FILE__, __LINE__);

  hipFree((void *) d_table);
  cudaCheckErrors("device freeing of GOL matrix failed", __FILE__, __LINE__);

  /******************************************************************************
   *                           Calculation execution                            *
   ******************************************************************************/

  // calculate iterations of game of life with GPU
  uint *d_tiled_help; /* Tiled help matrix in device memory. */
  hipMalloc((void **) &d_tiled_help, mem_size_tiled);
  cudaCheckErrors("device allocation of help matrix failed", __FILE__, __LINE__);

  // start timewatch
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //TODO: synchronize here?
  for (int i = 0; i < n_runs; ++i) {
    calculate_next_generation <<< grid, block >>>(
      d_tiled_table, d_tiled_help, m_height, m_width, total_elements_tiled);
    cudaCheckErrors("calculating next generation failed", __FILE__, __LINE__);
    swap_uint(&d_tiled_table, &d_tiled_help);
  }

  // end timewatch
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("CUDA time to run:  %f s \n", time / 1000);

  hipFree((void *) d_tiled_help);
  cudaCheckErrors("device freeing of help matrix failed", __FILE__, __LINE__);

  /******************************************************************************
   *                  Device finalization and Result printing                   *
   ******************************************************************************/

  // allocation again of a matrix that holds the normal representation
  hipMalloc((void **) &d_table,  mem_size);
  cudaCheckErrors("device allocation of GOL matrix failed", __FILE__, __LINE__);

  // convert back to normal representation of the matrix
  convert_from_tiled <<< grid, block >>>(d_table, d_tiled_table,
                                         m_height, m_width, total_elements_tiled);
  cudaCheckErrors("failed to convert to normal repr from uint tiled repr", __FILE__, __LINE__);

  // transfer memory from resulting matrix from device to host
  hipMemcpy(table, d_table, mem_size, hipMemcpyDeviceToHost);
  cudaCheckErrors("copy from device to host memory failed", __FILE__, __LINE__);

  // reset gpu
  hipDeviceReset();

  // save results to a file
  save_table(table, dim, dim, "cuda-2-results.bin");
#ifdef PRINT
  print_table(table, dim, dim);
#endif  // PRINT
  free((void *) table);
}
#endif  // TESTING
